
#include <hip/hip_runtime.h>
#include <omp.h>
#include <vector>
#include <fstream>
#include <iostream>

#define	INFTY	1e8

using namespace std;

__global__ void calcH(int V, int mid, int *g, int *h)
{
  int start = threadIdx.y + blockIdx.y * blockDim.y;
  int end   = threadIdx.x + blockIdx.x * blockDim.x;
  h[start*V + end] = min(g[start*V + end], g[start*V + mid] + g[mid*V + end]);
}

__global__ void calcG(int V, int *g, int *h)
{
  int start = threadIdx.y + blockIdx.y * blockDim.y;
  int end   = threadIdx.x + blockIdx.x * blockDim.x;
  g[start*V + end] = h[start*V + end];
}

int main(int argv, char **argc)
{
  int V = strtol(argc[1], (char **)NULL, 10), E = 0;
  int *g, *h;
  
  hipMallocManaged(&g, V*V*sizeof(int));
  hipMallocManaged(&h, V*V*sizeof(int));
  
  for (int start = 0; start < V; start++)
    for (int end = 0; end < V; end++)
      if (rand()%((V*V)/(3*V)))
	  {
	    g[start*V + end] = rand()%500 + 1;
        E++;
      }
      else
        g[start*V + end] = INFTY;
  
  int threads = 10 * 10;
  int blocks  = (V + threads - 1) / threads;
  dim3 THREADS (threads, threads);
  dim3 BLOCKS  ( blocks,  blocks);
  
  for (int mid = 0; mid < V; mid++)
  {
    calcH<<<BLOCKS, THREADS>>>(V, mid, g, h);
    hipDeviceSynchronize();
    calcG<<<BLOCKS, THREADS>>>(V, g, h);
    hipDeviceSynchronize();
  }
  
  printf("Time taken for CUDA implementation with (V = \t%d) = ", V);
  return 0;
}