#include "hip/hip_runtime.h"
#include <omp.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <bits/stdc++.h>

#define INFTY   1e8

#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif 

using namespace std;

class Graph
{
	int V;
public:
	list< pair<int, int> > *adj;
	vector<int> path;

	Graph(int V);
	int vertNum();
	void addEdge(int u, int v, int w);
	void modifyEdge(int u, int v, int w);
	void Dijkstras_Algo(int s, int dest);
	void BellmanFord_Algo(int s);
	void FloydWarshall_Algo(int s);
};

int Graph::vertNum()
{
	return this->V;
}
Graph::Graph(int V)
{
	this->V = V;
	adj = new list< pair<int, int> >[V];
}
void Graph::addEdge(int u, int v, int w)
{
	adj[u].push_back(make_pair(v, w));
	adj[v].push_back(make_pair(u, w));
}
void Graph::modifyEdge(int u, int v, int w)
{
	list< pair<int, int> >::iterator it;
	for (it = this->adj[u].begin(); it != this->adj[u].end(); it++)
	{
		if ((*it).first == v)
			(this->adj)[u].erase(it);
		break;
	}
	for (it = this->adj[v].begin(); it != this->adj[v].end(); it++)
	{
		if ((*it).first == u)
			(this->adj)[v].erase(it);
		break;
	}
	
	adj[u].push_back(make_pair(v, w));
	adj[v].push_back(make_pair(u, w));
}

__global__ void matrixMul(Graph g)
{
  for(int mid = 0; mid < n_v; mid++)
	{
		for (int i = 0; i < n_v*n_v; i++)
		{			
				list< pair<int, int> >::iterator it;
				int w1=1e8;
				int w2=1e8;
				int w3=1e8;
				
				int start = i / n_v;
				int end = i % n_v;
				for (it = g.adj[start].begin(); it != g.adj[start].end(); it++)
				{
					if ((*it).first == end)
					(*it).second = w1;
					if((*it).first == mid)
					(*it).second = w2;
				}
				
				for (it = g.adj[mid].begin(); it != g.adj[mid].end(); it++)
				{
					if ((*it).first == end)
					(*it).second = w3;

				}
				
				int w = min(w1, w2+w3);
				h.modifyEdge(start,end,w);

		}

		#pragma omp parallel for shared(g)
		for(int i=0; i < n_v; i++)
		{
			list< pair<int, int> >::iterator it;
			for(it = h.adj[i].begin(); it != dis.adj[i].end(); it++)
			{
				g.modifyEdge(i, (*it).first, (*it).second);
			}
			
		}
	}
	
}

int main(int argv, char **argc)
{
	int V = strtol(argc[1], (char **)NULL, 10), E = 3*V;
	
	Graph g(V);
	Graph h(V);
	g.addEdge(0,2,1);
	for (int j = 1; j < E; j++) {
		g.addEdge(rand()%(V) + 0, rand()%(V) + 0, rand()%(50 - 1 + 1) + 1);
	}


	int numThreads = strtol(argc[2], (char **)NULL, 10), threadNum;
	omp_set_num_threads(numThreads);

	int n_v = g.vertNum();
	
  int threads = 10 * 10;
  int blocks  = (N + threads - 1) / threads;
  double t1 = omp_get_wtime();
  dim3 THREADS (threads, threads);
  dim3 BLOCKS  ( blocks,  blocks);
	matrixMul<<<BLOCKS, THREADS>>>(g);
	hipDeviceSynchronize();
  double t2 = omp_get_wtime() - t1;
	cout<<"Total Time taken for \t"<<V<<" vertices = "<<t2<<" seconds."<<endl;

	return 0;
}
