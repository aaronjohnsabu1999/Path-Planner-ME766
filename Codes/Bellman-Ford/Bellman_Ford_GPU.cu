
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <stdio.h>
#include <climits>
#include <random>
#include <fstream>
#include <iostream>
#include <vector>

using namespace std;
  
struct Edge 
{
    int src, dest, weight;
};
    
__global__ void SetupDist(int V, int *dist, int src) 
{	
	// Initialize Distances as INT_Max
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < V; i += stride)
        dist[i] = INT_MAX;
    dist[src] = 0;
}
  
__global__ void BellmanFord(int V, int E, struct Edge* edges, int *dist) {
  // Kernel with Grid Stride Loop
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
 // Step 2: Relax all edges |V| - 1 times
  for (int j = index; j < E; j += stride) {
    int u      = edges[j].src;
    int v      = edges[j].dest;
    int weight = edges[j].weight;

 // Atomic to avoid two threads writing simultaneously
	if (dist[u] != INT_MAX && dist[u] + weight < dist[v])
	  atomicMin(&dist[v], dist[u] + weight);
  }
  return;
}
  
__global__ void Check_Neg_Cycle(int E, struct Edge* edges, int *dist) 
{

  int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int flag = 1;
    for (int i = index; i < E; i += stride) {
        int u = edges[i].src;
        int v = edges[i].dest;
        int weight = edges[i].weight;
        if (dist[u] != INT_MAX && dist[u] + weight < dist[v]) {
            flag = -1;
        }
    }
    if(flag == -1)
        printf("Graph contains negative weight cycle");

return;
}

// Driver program to test above functions
int main()
{
  vector<vector<int>> vect;
    fstream disFile;
    disFile.open("/home/170100094/USA-road-d_NY.txt");

    if (disFile.is_open()){ 
        string tp;

        while(getline(disFile, tp)){ 
            vector<int> pair;
            if(tp[0] == 'a'){
                int a;
                string str(tp.begin()+1, tp.begin()+tp.size() );
                stringstream ss (str);
                while ((ss >> a))
                    pair.push_back (a);
                vect.push_back(pair);
            }
        }

        disFile.close(); 
   }

    int V = 264346; // Number of vertices in graph
    int E = vect.size(); // Number of edges in graph

    int *host_dist;
    Edge *host_edges;
    host_edges    = (Edge*)malloc(E*sizeof(Edge));
    host_dist     = (int*)malloc(V*sizeof(int));

	Edge *gpu_edges;
	int *gpu_dist;

	// allocate memory
	hipMalloc(&gpu_edges, E * sizeof(Edge));
	hipMalloc(&gpu_dist, V * sizeof(int));

    for (int j = 1; j < E; j++) 
    {
    host_edges[j].src    = vect[j][0]-1;
    host_edges[j].dest   = vect[j][1]-1;
    host_edges[j].weight = vect[j][2];
  	}
  
    cout<<host_edges[10].weight;
	
	// copy over to device/GPU
	hipMemcpy(gpu_edges, host_edges, E*sizeof(Edge), hipMemcpyHostToDevice);
	hipMemcpy(gpu_dist, host_dist, V*sizeof(int), hipMemcpyHostToDevice);

	// Setup threads
    int blockSize = 256;
    int numBlocksE = (E + blockSize - 1) / blockSize;

    int numBlocksV = (V + blockSize - 1) / blockSize;

    SetupDist<<<numBlocksV, blockSize>>>(V, gpu_dist, 0);
    hipDeviceSynchronize();

    for (int i = 1; i <= V - 1; i++) 
        {
            BellmanFord<<<numBlocksE, blockSize>>>(V, E, gpu_edges, gpu_dist);
            hipDeviceSynchronize();
        }

Check_Neg_Cycle<<<numBlocksE, blockSize>>>(E, gpu_edges, gpu_dist);
hipDeviceSynchronize();

  // Copy results to host
  hipMemcpy(host_dist, gpu_dist, V*sizeof(int), hipMemcpyDeviceToHost);

  // cleanup memory
  hipFree(gpu_edges);
  hipFree(gpu_dist);
  free(host_dist);
  free(host_edges);
  
  return 0;
}
