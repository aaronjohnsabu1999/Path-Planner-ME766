#include "hip/hip_runtime.h"
#include <omp.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <bits/stdc++.h>

#define INFTY   1e8

using namespace std;

class Graph
{
	int V;
public:
	list< pair<int, int> > *adj;
	vector<int> path;

	Graph(int V);
	int vertNum();
	void addEdge(int u, int v, int w);
	void modifyEdge(int u, int v, int w);
};

int Graph::vertNum()
{
	return this->V;
}
Graph::Graph(int V)
{
	this->V = V;
	adj = new list< pair<int, int> >[V];
}
void Graph::addEdge(int u, int v, int w)
{
	adj[u].push_back(make_pair(v, w));
	adj[v].push_back(make_pair(u, w));
}
void Graph::modifyEdge(int u, int v, int w)
{
	list< pair<int, int> >::iterator it;
	for (it = this->adj[u].begin(); it != this->adj[u].end(); it++)
	{
		if ((*it).first == v)
			(this->adj)[u].erase(it);
		break;
	}
	for (it = this->adj[v].begin(); it != this->adj[v].end(); it++)
	{
		if ((*it).first == u)
			(this->adj)[v].erase(it);
		break;
	}
	
	adj[u].push_back(make_pair(v, w));
	adj[v].push_back(make_pair(u, w));
}

__global__ void matrixMul(int mid, vector< pair<int,int> > xMid, Graph g)
{
  for (int start = 0; start < g.vertNum(); start++)
	{
    vector< pair<int,int> > xStart;
		xStart.clear();
		xStart.reserve(g.adj[start].size());
		copy(g.adj[start].begin(), g.adj[start].end(), xStart.begin());
		
		for (end = 0; end < g.vertNum(); end++)
		{
			vector< pair<int,int> > xEnd;
			xEnd.clear();
			xEnd.reserve(g.adj[end].size());
			copy(g.adj[end].begin(), g.adj[end].end(), xEnd.begin());
		
			int  posStartEnd = -1, posStartMid = -1, posMidEnd = -1;
			int  valStartEnd,      valStartMid,      valMidEnd;
			int doneStartEnd,     doneStartMid,     doneMidEnd;
			
			for (auto itStart = xStart.begin(); itStart != xStart.end(); ++itStart)
			{
				if (doneStartEnd == 0)
				{
					posStartEnd += 1;
					valStartEnd = (*itStart).second;
				}
				if (doneStartMid == 0)
				{
					posStartMid += 1;
					valStartMid = (*itStart).second;
				}
				if ((*itStart).first == end)
					doneStartEnd = 1;
				if ((*itStart).first == mid)
					doneStartMid = 1;
				if (doneStartEnd == 1 && doneStartMid == 1)
					break;
			}
			
			for (auto itMid = xMid.begin(); itMid != xMid.end(); ++itMid)
			{
				if (doneMidEnd == 0)
				{
					posMidEnd += 1;
					valMidEnd = (*itMid).second;
				}
				if ((*itMid).first == end)
					doneMidEnd = 1;
				if (doneMidEnd == 1)
					break;
			}
			pair<int,int> replaceNode = make_pair(posStartEnd, valStartEnd);
			(g.adj)[start].remove(replaceNode);
			(g.adj)[start].push_back(make_pair(posStartEnd, min(valStartEnd, valStartMid + valMidEnd)));
		}
	}
  
  int start = threadIdx.y + blockIdx.y * blockDim.y;
  int start = threadIdx.x + blockIdx.x * blockDim.x;
  int k;
  
  if (i < N && j < N)
  {
    float temp = 0;
    for (k = 0; k < N; k++)
      temp += A[i*N + k] * B[k*N + j];
    C[i*N + k] = temp;
  }
}

int main(int argv, char **argc)
{
	/*
	fstream disFile;
	disFile.open("./USA-road-d_NY.txt");
	
	vector< vector<int> > vect;
	if (disFile.is_open()){ 
		string tp;
		while(getline(disFile, tp)){ 
			vector<int> pair;
			if(tp[0] == 'a'){
				int a;
				string str(tp.begin()+1, tp.begin()+tp.size() );
				stringstream ss (str);
				while ((ss >> a))
					pair.push_back (a);
				vect.push_back(pair);
			}
		}
		disFile.close(); 
	}
	*/
    
	int V = strtol(argc[1], (char **)NULL, 10), E = 3*V;
	Graph g(V);
	g.addEdge(0,2,1);
	for (int j = 1; j < E; j++) {
		g.addEdge(rand()%(V) + 0, rand()%(V) + 0, rand()%(50 - 1 + 1) + 1);
	}
	/*
	// Code for importing NYC Road Map -- Impossible to solve in a lifetime
	Graph g(264346);
	for (int i = 0; i < vect.size(); i++) {
		g.addEdge(vect[i][0]-1,vect[i][1]-1,vect[i][2]);
	}
	cout<<"Number of edges: "<<vect.size()<<'\n';
	*/
	
	// Floyd-Warshall Algorithm
	int numThreads = strtol(argc[2], (char **)NULL, 10), threadNum;
	int start, end, mid;
	
	double t1 = omp_get_wtime();
	for (mid = 0; mid < g.vertNum(); mid++)
    {
		double tM1 = omp_get_wtime();
		
		vector< pair<int,int> > xMid;
		xMid.clear();
		xMid.reserve(g.adj[mid].size());
		copy(g.adj[mid].begin(), g.adj[mid].end(), xMid.begin());
		
		
   }
	// double t2 = omp_get_wtime() - t1;
	// cout<<"Total Time taken for \t"<<V<<" vertices = "<<t2<<" seconds."<<endl;
	
	return 0;
}